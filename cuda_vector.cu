#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 1000000 // Size of the vectors (adjust as needed)

__global__ void sequential_vector_add(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

__global__ void parallel_vector_add(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    // Allocate host memory
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));

    // Initialize host arrays (example with random values)
    for (int i = 0; i < N; i++) {
        h_a[i] = rand();
        h_b[i] = rand();
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Grid and block sizes for parallel execution
    int threadsPerBlock = 256;  // Adjust as needed based on GPU architecture
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  float sequential_time, parallel_time;
    // Measure execution times using a single loop
    hipEvent_t start_event, end_event;
    hipEventCreate(&start_event);
    hipEventCreate(&end_event);

    // Sequential execution
    hipEventRecord(start_event, 0);
    for (int i = 0; i < 100; ++i) {  // Run the kernel multiple times for better timing
        sequential_vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    }
    hipDeviceSynchronize();
    hipEventRecord(end_event, 0);
    hipEventElapsedTime(&sequential_time, start_event, end_event);
    hipEventDestroy(start_event);

    // Parallel execution
    hipEventRecord(start_event, 0);
    for (int i = 0; i < 100; ++i) {  // Run the kernel multiple times for better timing
        parallel_vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    }
    hipDeviceSynchronize();
    hipEventRecord(end_event, 0);
    hipEventElapsedTime(&parallel_time, start_event, end_event);
    hipEventDestroy(end_event);
    sequential_time /= 100.0f;  // Average time over multiple runs
    parallel_time /= 100.0f;

    // Copy results back from device to host (optional for verification)
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Calculate speedup
    float speedup = sequential_time / parallel_time;

    printf("Sequential execution time: %.6f ms\n", sequential_time);
    printf("Parallel execution time: %.6f ms\n", parallel_time);
        printf("Speedup: %.2f\n", speedup);


    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    return 0;
    }

